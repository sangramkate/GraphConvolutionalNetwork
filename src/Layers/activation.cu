#include "hip/hip_runtime.h"
#include "activation.hh"
#include "nn_exception.hh"
#include <iostream>
__global__ void ReluActivationForward(float* Z, float* A,float* Stored_Z, int Z_x_dim, int Z_y_dim) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < Z_x_dim * Z_y_dim) {
		A[index] = fmaxf(Z[index], 0);
                Stored_Z[index] = A[index];
	}
}

__global__ void ReluActivationBackprop(float* Z, float* dA, float* dZ, int Z_x_dim, int Z_y_dim) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < Z_x_dim * Z_y_dim) {
		if (Z[index] > 0) {
			dZ[index] = dA[index];
		}
		else {
			dZ[index] = 0;
		}
	}
}

ReLUActivation::ReLUActivation(std::string name) {
	this->name = name;
}

ReLUActivation::~ReLUActivation() { }

Matrix& ReLUActivation::forward(Matrix& P, bool training, bool freeMatrix) {
        //std::cout << "Relu Layer forward\n";
	this->Z = P;
	A.allocateCuda(Z.shape); 
	stored_Z.allocateCuda(Z.shape);

	dim3 block_size(64);
	dim3 num_of_blocks((Z.shape.y * Z.shape.x + block_size.x - 1) / block_size.x);

	ReluActivationForward<<<num_of_blocks, block_size>>>(Z.data_device, A.data_device,stored_Z.data_device, Z.shape.x, Z.shape.y);
       // std::cout << "Relu forward\n";
	NNException::throwIfDeviceErrorOccurred("Cannot perform ReLU forward propagation.");
	//printf("P.x %d P.y %d\n",Z.shape.x, Z.shape.y);
	//printf("relu out A.x %d A.y %d\n",A.shape.x, A.shape.y);
        P.freeMem();
	//printf("here?\n");
        if(training == false){
           stored_Z.freeMem();
        }
	return A;
}

Matrix& ReLUActivation::backprop(Matrix& dA, float learning_rate) {
        //std::cout << "Relu Layer backward\n";
	dZ.allocateCuda(stored_Z.shape);
	//printf("relu in back dA.shape %d dA.shape %d\n",dA.shape.x, dA.shape.y);
	dim3 block_size(64);
	dim3 num_of_blocks((stored_Z.shape.y * stored_Z.shape.x + block_size.x - 1) / block_size.x);
	ReluActivationBackprop<<<num_of_blocks, block_size>>>(stored_Z.data_device, dA.data_device,dZ.data_device, stored_Z.shape.x, stored_Z.shape.y);
        //std::cout << "Relu Backward\n"; 	
        NNException::throwIfDeviceErrorOccurred("Cannot perform ReLU back propagation");
	//printf("relu out back dZ.shape %d dA.shape %d\n",dZ.shape.x, dZ.shape.y);

        dA.freeMem();
        stored_Z.freeMem();
	return dZ;
}

void ReLUActivation::setData(int* row, int* col) {
}
