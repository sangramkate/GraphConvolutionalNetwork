#include <iostream>
#include <time.h>
#include <fstream>
#include <string>
#include <stdlib.h>

#include "NeuralNetwork.hh"
#include "linear_layer.hh"
#include "activation.hh"
#include "softmax.hh"
#include "nodeaggregator.hh"
#include "nn_exception.hh"
#include "costfunction.hh"
#include "csr_graph.h"
#include "Layers/csr_graph.cu"
#include "data.hh" 
#include <chrono> 
using namespace std::chrono;

float computeAccuracy(const Matrix& predictions, const Matrix& targets);

int main() {

        std::fstream myfile("/net/ohm/export/iss/inputs/Learning/cora-labels.txt", std::ios_base::in);
	int nodes = 2708;
	int label_size = 7;
        int* label = (int *) malloc(nodes*label_size*sizeof(int));
        int i = 0;
        int a;
        myfile >> a;
        myfile >> a;
        while (myfile >> a)
        {
            label[i] = a;
            i++;
        }
        myfile.close();

	srand( time(NULL) );

	//CoordinatesDataset dataset(100, 21);
	CostFunction bce_cost;

//Code for extracting data from dataset files starts here
        CSRGraph graph;
        char gr_file[]="cora.gr";
        char binFile[]="cora-feat.bin";
        int feature_size = 1433;
	int edges = 5278;
        graph.read(gr_file);
        int* d_row_start;
        int* d_row_start_test;
        int* d_edge_dst;
        int* d_edge_dst_test;
        float* d_edge_data;
        hipError_t alloc;
        int nnz = 5278;
        alloc = hipMalloc(&d_row_start_test,(nodes+1) * sizeof(int));
        if(alloc != hipSuccess) {
            printf("malloc for row info failed\n");
        }
        alloc = hipMalloc(&d_edge_dst_test,(edges+nodes) * sizeof(int));
        if(alloc != hipSuccess) {
            printf("malloc for col info failed\n");
        }
        alloc = hipMalloc(&d_row_start,(nodes+1) * sizeof(int));
        if(alloc != hipSuccess) {
            printf("malloc for row info failed\n");
        }
        alloc = hipMalloc(&d_edge_dst,(edges+nodes) * sizeof(int));
        if(alloc != hipSuccess) {
            printf("malloc for col info failed\n");
        }
        float* d_B;

        float* h_B = (float *)malloc((nodes) * feature_size * sizeof(float));
	if(h_B == NULL)
	    printf("h_B malloc failed\n");
        alloc = hipMalloc(&d_B, (nodes) * feature_size * sizeof(float));
        if(alloc != hipSuccess) {
            printf("hipMalloc failed for features matrix\n");
        }
        alloc = hipMalloc(&d_edge_data,(nnz+nodes) * sizeof(float));
        if(alloc != hipSuccess) {
            printf("malloc failed \n");
        }
	float* h_edge_data = (float *)malloc((nnz+nodes) * sizeof(float));
        for(int i=0;i<(nnz+nodes);i++)
            h_edge_data[i] = 1.0;
	alloc = hipMemcpy(d_edge_data, h_edge_data, ((nnz+nodes) *sizeof(float)), hipMemcpyHostToDevice);
        if(alloc != hipSuccess) {
        printf("Feature matrix memcpy failed\n");
        }

//Filling up the sparse matrix info
        graph.readFromGR(gr_file , binFile , d_row_start, d_edge_dst , d_row_start_test, d_edge_dst_test, d_B, feature_size);
        alloc = hipMemcpy(h_B, d_B, (nodes * feature_size *sizeof(float)), hipMemcpyDeviceToHost);
	if(alloc != hipSuccess) {
    	printf("Feature matrix memcpy failed\n");
	} 
	std::cout << "Dataset captured!\n";
        Data dataset(2708,100,feature_size,label_size,label,h_B);
      //  free(label);
       // free(h_B);
	Shape input_shape(140,feature_size);
	Matrix input;
        input.allocateMemoryIfNotAllocated(input_shape);
	SpMM(d_edge_data, d_row_start, d_edge_dst, d_B, input.data_device, feature_size, 140, 756);
	std::cout << "Dataset captured!\n";
        NeuralNetwork nn(0.001);
        //-----------------------------------------------
        std::cout << "Instance of Neural Network\n";
//	nn.addLayer(new NodeAggregator("nodeagg1", d_edge_data, d_row_start, d_edge_dst, 1000, 4132));
        std::cout << "Added Nodeaggregator 1 layer\n";
	nn.addLayer(new LinearLayer("linear1", Shape(label_size,feature_size)));
        std::cout << "Added Linear layer 1\n";
	nn.addLayer(new ReLUActivation("relu1"));
        std::cout << "Added relu layer 1\n";
        //-----------------------------------------------
       // nn.addLayer(new NodeAggregator("nodeagg2", d_edge_data, d_row_start, d_edge_dst, 2708, nnz));
       // std::cout << "Added Nodeaggregator layer 2\n";
       // nn.addLayer(new LinearLayer("linear2", Shape(label_size,label_size)));
       // std::cout << "Added Linear layer 2\n";
       // nn.addLayer(new ReLUActivation("relu2"));
       // std::cout << "Added Relu layer 2\n"; 
        //-----------------------------------------------
    //    nn.addLayer(new NodeAggregator("nodeagg3", d_edge_data, d_row_start, d_edge_dst, 2708, nnz));
    //    std::cout << "Added Nodeaggregator layer 3\n";
    //    nn.addLayer(new LinearLayer("linear3", Shape(label_size,label_size)));
    //    std::cout << "Added Linear layer 3\n";
    //    nn.addLayer(new ReLUActivation("relu3"));
    //    std::cout << "Added Relu layer 3\n"; 
        //-----------------------------------------------
        nn.addLayer(new SoftMax("softmax"));
        std::cout << "Added softmax layer \n";
	nn.getLayers();
        std::cout << "Instance of Neural Network complete\n";
	// network training
	nn.NodeAggSetData(d_row_start_test, d_edge_dst_test);
	//NodeAggregator.setData(d_row_start, d_edge_dst);
	Matrix Y;
	for (int epoch = 0; epoch < 1001; epoch++) {
		float cost = 0.0;
		auto start = high_resolution_clock::now();
//		for (int batch = 0; batch < dataset.getNumOfTrainingBatches(); batch++) {
                       // std::cout << "input_features:" << dataset.input_features.data_device << "\n";
			Y = nn.forward(input, true);
			//Y = nn.forward(dataset.input_features, true);
			nn.backprop(Y,dataset.input_labels);
                        //std::cout << "cost computation start \n";
			cost += bce_cost.cost(Y,dataset.input_labels);
			auto stop = high_resolution_clock::now();
                        //std::cout << "cost computed!\n";
//		}
			auto duration = duration_cast<microseconds>(stop - start);
//                std::cout << "epoch:" << epoch << "\n";
		if (epoch % 100 == 0) {
			std::cout 	<< "Epoch: " << epoch
						<< ", Cost: " << cost / 100 << " " << "Duration: " << duration.count() << "\n";
		}
                Y.freeMem();
	}

        float accuracy = 0.0f;
        float final_accuracy = 0.0f;
//	for (int batch = 0; batch < dataset.getNumOfTestBatches(); batch++) {
		Y = nn.forward(input, false);
		//Y = nn.forward(dataset.input_features, false);
                Y.allocateHostMemory();
                std::cout << "Y.host allocated:" << Y.host_allocated << "\n";
		Y.copyDeviceToHost();
                std::cout << "Y copied to host "<< "\n";
   //             accuracy = accuracy + computeAccuracy(Y,h_B);
//	}
	int m = Y.shape.x * Y.shape.y;
	int correct_predictions = 0;

	for (int i = 0; i < m; i++) {
		float prediction = Y[i] > 0.5 ? 1 : 0;
		if (prediction == label[i]) {
			correct_predictions++;
		}
	}
	final_accuracy =  static_cast<float>(correct_predictions) / m;
  //      final_accuracy = accuracy;
	// compute accuracy
        
	std::cout << "Accuracy: " << final_accuracy << std::endl;
        hipFree(d_row_start);
        hipFree(d_edge_dst);
        hipFree(d_row_start_test);
        hipFree(d_edge_dst_test);
        hipFree(d_B);
        hipFree(d_edge_data);
        dataset.input_features.freeMem();
        dataset.input_labels.freeMem();
	free(label);
	return 0;

}
float computeAccuracy(const Matrix& predictions, const Matrix& targets) {
	int m = predictions.shape.x * predictions.shape.y;
	int correct_predictions = 0;

	for (int i = 0; i < m; i++) {
		float prediction = predictions[i] > 0.5 ? 1 : 0;
		if (prediction == targets[i]) {
			correct_predictions++;
		}
	}
	return static_cast<float>(correct_predictions) / m;
}
