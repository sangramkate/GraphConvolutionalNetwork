#include <iostream>
#include <time.h>

#include "NeuralNetwork.hh"
#include "linear_layer.hh"
#include "activation.hh"
#include "nodeaggregator.hh"
#include "nn_exception.hh"
#include "costfunction.hh"
#include "csr_graph.h"
float computeAccuracy(const Matrix& predictions, const Matrix& targets);

int main() {

	srand( time(NULL) );

	//CoordinatesDataset dataset(100, 21);
	CostFunction bce_cost;

//Code for extracting data from dataset files starts here
        CSRGraph graph;
        char gr_file[]="cora.gr";
        char binFile[]="cora-feat.bin";
        int *nnodes = 0,*nedges = 0;
        int feature_size = 1433;
        graph.read(gr_file,nnodes,nedges);
        int* d_row_start;
        int* d_edge_dst;
        float* d_edge_data;
        hipError_t alloc;
        int nnz = *nedges;
        alloc = hipMalloc(&d_row_start,(*nnodes+1) * sizeof(*d_row_start));
        if(alloc != hipSuccess) {
            printf("malloc for row info failed\n");
        }
        alloc = hipMalloc(&d_edge_dst,(*nedges) * sizeof(*d_edge_dst));
        if(alloc != hipSuccess) {
            printf("malloc for col info failed\n");
        }
        float* d_B;
        alloc = hipMalloc(&d_B, (*nnodes) * feature_size * sizeof(float));
        if(alloc != hipSuccess) {
            printf("hipMalloc failed for features matrix\n");
        }
        alloc = hipMalloc(&d_edge_data,nnz * sizeof(*d_edge_data));
        if(alloc != hipSuccess) {
            printf("malloc failed \n");
        }
        alloc = hipMemset(d_edge_data, 1, *nedges*sizeof(float));
        if(alloc != hipSuccess) {
            printf("memset for edge data failed \n");
        }
//Filling up the sparse matrix info
        graph.readFromGR(gr_file , binFile , d_row_start, d_edge_dst , d_B, feature_size);

	NeuralNetwork nn;

	nn.addLayer(new NodeAggregator("nodeagg1", d_edge_data, d_row_start, d_edge_dst, *nnodes, nnz));
	nn.addLayer(new LinearLayer("linear1", Shape(100,20)));
	nn.addLayer(new ReLUActivation("relu2"));
	nn.addLayer(new NodeAggregator("nodeagg2", d_edge_data, d_row_start, d_edge_dst, *nnodes, nnz));
	nn.addLayer(new LinearLayer("linear2", Shape(100,20)));
	nn.addLayer(new ReLUActivation("relu2"));
        nn.addLayer(new SoftMax("softmax"));

	// network training
	Matrix Y;
	for (int epoch = 0; epoch < 1001; epoch++) {
		float cost = 0.0;

		for (int batch = 0; batch < 100 - 1; batch++) {
			Y = nn.forward(Y);
			nn.backprop(Y,);
			cost += bce_cost.cost(Y,);
		}

		if (epoch % 100 == 0) {
			std::cout 	<< "Epoch: " << epoch
						<< ", Cost: " << cost / 100
						<< std::endl;
		}
	}

	// compute accuracy
	Y = nn.forward();
	Y.copyDeviceToHost();

	float accuracy = computeAccuracy(Y,);
	std::cout << "Accuracy: " << accuracy << std::endl;

	return 0;
}

float computeAccuracy(const Matrix& predictions, const Matrix& targets) {
	int m = predictions.shape.x;
	int correct_predictions = 0;

	for (int i = 0; i < m; i++) {
		float prediction = predictions[i] > 0.5 ? 1 : 0;
		if (prediction == targets[i]) {
			correct_predictions++;
		}
	}
	return static_cast<float>(correct_predictions) / m;
}
